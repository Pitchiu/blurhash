#include "hip/hip_runtime.h"

#include <string.h>

#include "encode_cpu.cuh"
#include "../common/stb_image.h"
#include "../common/common.h"


void encodeCPU(const ProgramOptions& options)
{
    int xComponents = options.xComponents;
	int yComponents = options.yComponents;

	for (auto const& filename : options.images)
	{
		std::string path(options.path + filename);
	    const char *hash = blurHashForFile(xComponents, yComponents, path.c_str());

		if (!hash)
		{
			fprintf(stderr, "Failed to load image file \"%s\".\n", path.c_str());
			exit(EXIT_FAILURE);
		}
		printf("%s\n", hash);
	}
}

const char *blurHashForFile(int xComponents, int yComponents,const char *filename)
{
	int width, height, channels;
	unsigned char *data = stbi_load(filename, &width, &height, &channels, 3);
	if(!data) return NULL;

	const char *hash = blurHashForPixels(xComponents, yComponents, width, height, data, width * 3);

	stbi_image_free(data);

	return hash;
}

const char *blurHashForPixels(int xComponents, int yComponents, int width, int height, uint8_t *rgb, size_t bytesPerRow) {
	static char buffer[2 + 4 + (9 * 9 - 1) * 2 + 1];

	float factors[yComponents][xComponents][3];
	memset(factors, 0, sizeof(factors));

	for(int y = 0; y < yComponents; y++) {
		for(int x = 0; x < xComponents; x++) {
			float *factor = multiplyBasisFunction(x, y, width, height, rgb, bytesPerRow);
			factors[y][x][0] = factor[0];
			factors[y][x][1] = factor[1];
			factors[y][x][2] = factor[2];
		}
	}

	float *dc = factors[0][0];
	float *ac = dc + 3;
	int acCount = xComponents * yComponents - 1;
	char *ptr = buffer;

	int sizeFlag = (xComponents - 1) + (yComponents - 1) * 9;
	ptr = encodeInt(sizeFlag, 1, ptr);

	float maximumValue;
	if(acCount > 0) {
		float actualMaximumValue = 0;
		for(int i = 0; i < acCount * 3; i++) {
			actualMaximumValue = fmaxf(fabsf(ac[i]), actualMaximumValue);
		}

		int quantisedMaximumValue = fmaxf(0, fminf(82, floorf(actualMaximumValue * 166 - 0.5)));
		maximumValue = ((float)quantisedMaximumValue + 1) / 166;
		ptr = encodeInt(quantisedMaximumValue, 1, ptr);
	} else {
		maximumValue = 1;
		ptr = encodeInt(0, 1, ptr);
	}

	ptr = encodeInt(encodeDC(dc[0], dc[1], dc[2]), 4, ptr);

	for(int i = 0; i < acCount; i++) {
		ptr = encodeInt(encodeAC(ac[i * 3 + 0], ac[i * 3 + 1], ac[i * 3 + 2], maximumValue), 2, ptr);
	}

	*ptr = 0;

	return buffer;
}

float *multiplyBasisFunction(int xComponent, int yComponent, int width, int height, uint8_t *rgb, size_t bytesPerRow) {
	float r = 0, g = 0, b = 0;
	float normalisation = (xComponent == 0 && yComponent == 0) ? 1 : 2;

	for(int y = 0; y < height; y++) {
		for(int x = 0; x < width; x++) {
			float basis = cosf(M_PI * xComponent * x / width) * cosf(M_PI * yComponent * y / height);
			r += basis * sRGBToLinear(rgb[3 * x + 0 + y * bytesPerRow]);
			g += basis * sRGBToLinear(rgb[3 * x + 1 + y * bytesPerRow]);
			b += basis * sRGBToLinear(rgb[3 * x + 2 + y * bytesPerRow]);
		}
	}

	float scale = normalisation / (width * height);

	static float result[3];
	result[0] = r * scale;
	result[1] = g * scale;
	result[2] = b * scale;

	return result;
}

int encodeAC(float r, float g, float b, float maximumValue) {
	int quantR = fmaxf(0, fminf(18, floorf(signCPUPow(r / maximumValue, 0.5) * 9 + 9.5)));
	int quantG = fmaxf(0, fminf(18, floorf(signCPUPow(g / maximumValue, 0.5) * 9 + 9.5)));
	int quantB = fmaxf(0, fminf(18, floorf(signCPUPow(b / maximumValue, 0.5) * 9 + 9.5)));

	return quantR * 19 * 19 + quantG * 19 + quantB;
}

float sRGBToLinear(int value) {
	float v = (float)value / 255;
	if(v <= 0.04045) return v / 12.92;
	else return powf((v + 0.055) / 1.055, 2.4);
}

float signCPUPow(float value, float exp) {
	return copysignf(powf(fabsf(value), exp), value);
}
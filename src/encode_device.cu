#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include "common.h"

#include "encode_device.cuh"

void encodeDevice(const ProgramOptions& options)
{
    int xComponents = options.xComponents;
	int yComponents = options.yComponents;

	int acCount = xComponents * yComponents - 1;
	float* deviceAc;
	CHECK_CUDA_ERROR(hipMalloc(&deviceAc, 3*acCount*sizeof(float)));

	
	char* encodedACValuesBuffer;
	CHECK_CUDA_ERROR(hipMalloc(&encodedACValuesBuffer, sizeof(char)*2*acCount));

	for (auto const& filename : options.images)
	{
		std::string path(options.path + filename);
		const char* hash = encodeFile(xComponents, yComponents, path.c_str(), deviceAc, encodedACValuesBuffer);

		if (!hash)
		{
			fprintf(stderr, "Failed to load image file \"%s\".\n", path.c_str());
			exit(EXIT_FAILURE);
		}
		printf("%s\n", hash);
	}
	CHECK_CUDA_ERROR(hipFree(deviceAc));
	CHECK_CUDA_ERROR(hipFree(encodedACValuesBuffer));
}

const char* encodeFile(int xComponents, int yComponents, const char* filename, float* deviceAc, char* encodedACValuesBuffer)
{
	int width, height, channels;

	unsigned char* data = stbi_load(filename, &width, &height, &channels, 3);
	if (!data) return NULL;

	const char* hash = encodePixels(data, xComponents, yComponents, width, height, deviceAc, encodedACValuesBuffer);
	stbi_image_free(data);
	return hash;
}


const char* encodePixels(uint8_t* data, int xComponents, int yComponents, int width, int height, float* deviceAc, char *encodedACValuesBuffer)
{
	static char buffer[2 + 4 + (9 * 9 - 1) * 2 + 1];

	float factors[yComponents * xComponents * 3];

	int N = width*height;
	uint8_t* pixels;
	CHECK_CUDA_ERROR(hipMalloc(&pixels, N*3*sizeof(uint8_t)));
	CHECK_CUDA_ERROR(hipMemcpy(pixels, data, N*3*sizeof(uint8_t), hipMemcpyHostToDevice));

	float* vec_r;
	float* vec_g;
	float* vec_b;

	CHECK_CUDA_ERROR(hipMalloc((void**)&vec_r, sizeof(float)*N));
	CHECK_CUDA_ERROR(hipMalloc((void**)&vec_g, sizeof(float)*N));
	CHECK_CUDA_ERROR(hipMalloc((void**)&vec_b, sizeof(float)*N));

	for (int y = 0; y < yComponents; y++) {
		for (int x = 0; x < xComponents; x++) {
			float factor[3];
			calculateFactor(pixels, x, y, width, height, factor, vec_r, vec_g, vec_b);
			factors[y * xComponents * 3 + x * 3 + 0] = factor[0];
			factors[y * xComponents * 3 + x * 3 + 1] = factor[1];
			factors[y * xComponents * 3 + x * 3 + 2] = factor[2];
		}
	}

	CHECK_CUDA_ERROR(hipFree(vec_r));
	CHECK_CUDA_ERROR(hipFree(vec_g));
	CHECK_CUDA_ERROR(hipFree(vec_b));
	CHECK_CUDA_ERROR(hipFree(pixels));


	float* dc = factors;
	float* ac = dc + 3;
	int acCount = xComponents * yComponents - 1;
	char* ptr = buffer;

	int sizeFlag = (xComponents - 1) + (yComponents - 1) * 9;
	ptr = encodeInt(sizeFlag, 1, ptr);
	float maximumValue;
	if (acCount > 0)
	{
		float actualMaximumValue = 0;
  		CHECK_CUDA_ERROR(hipMemcpy(deviceAc, ac, 3*acCount*sizeof(float), hipMemcpyHostToDevice));

		thrust::device_ptr<float> deviceAcPtr(deviceAc);
		actualMaximumValue = fabsf(*(thrust::max_element(deviceAcPtr, deviceAcPtr + 3*acCount, compare())));

		int quantisedMaximumValue = fmaxf(0, fminf(82, floorf(actualMaximumValue * 166 - 0.5)));
		maximumValue = ((float)quantisedMaximumValue + 1) / 166;
		ptr = encodeInt(quantisedMaximumValue, 1, ptr);
	}
	else
	{
		maximumValue = 1;
		ptr = encodeInt(0, 1, ptr);
	}

	ptr = encodeInt(encodeDC(dc[0], dc[1], dc[2]), 4, ptr);

	int numblocks = acCount / 128 + 1;
	encodeACValues<<<numblocks, 128>>>(deviceAc, encodedACValuesBuffer, maximumValue, acCount);
	hipDeviceSynchronize();
  	CHECK_CUDA_ERROR(hipMemcpy(buffer + 6, encodedACValuesBuffer, 2*acCount*sizeof(char), hipMemcpyDeviceToHost));

	return buffer;
}


void calculateFactor(uint8_t* pixels, int xComponent, int yComponent, int width, int height, float result[3], float* vec_r, float* vec_g, float* vec_b)
{
	float r = 0, g = 0, b = 0;
	float normalisation = (xComponent == 0 && yComponent == 0) ? 1 : 2;
	int N = width * height;

	int numblocks = N/1024 + 1;
	calculatePixelColors<<<numblocks, 1024>>>(pixels, vec_r, vec_g, vec_b, xComponent, yComponent, width, height);
	hipDeviceSynchronize();

	thrust::device_ptr<float> r_ptr(vec_r);
	thrust::device_ptr<float> g_ptr(vec_g);
	thrust::device_ptr<float> b_ptr(vec_b);

	r = thrust::reduce(r_ptr, r_ptr + N, (float) 0);
	g = thrust::reduce(g_ptr, g_ptr + N, (float) 0);
	b = thrust::reduce(b_ptr, b_ptr + N, (float) 0);

	float scale = normalisation / (width * height);

	result[0] = r * scale;
	result[1] = g * scale;
	result[2] = b * scale;
}